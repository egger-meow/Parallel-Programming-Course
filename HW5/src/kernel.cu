#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define THREADS_PER_BLOCK 256
#define GROUP_SIZE 2

__device__ int mandel(float c_re, float c_im, int maxIterations) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < maxIterations; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandelKernel (float lowerX, float lowerY, float stepX, float stepY, 
                             int maxIterations, int* img, size_t pitch, int resX, int resY)  {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int group = 0; group < GROUP_SIZE; ++group) {
        int idx = threadID * GROUP_SIZE + group;

        if (idx >= resX * resY)
            continue;

        int x = idx % resX;     
        int y = idx / resX;      

        float c_re = lowerX + x * stepX;
        float c_im = lowerY + y * stepY;

        int iter = mandel(c_re, c_im, maxIterations);

        int* row = (int*)((char*)img + y * pitch);
        row[x] = iter;
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int threadsPerBlockLocal = THREADS_PER_BLOCK;
    int blocksPerGrid = (resX * resY + threadsPerBlockLocal - 1) / threadsPerBlockLocal;


    int* hostImg;
    hipHostAlloc((void**)&hostImg, resX * resY * sizeof(int), hipHostMallocDefault);

    int* devImg;
    size_t pitch;
    hipMallocPitch((void**)&devImg, &pitch, resX * sizeof(int), resY);

    mandelKernel<<<blocksPerGrid, threadsPerBlockLocal>>>(
        lowerX, lowerY, stepX, stepY, maxIterations, devImg, pitch, resX, resY);

    hipDeviceSynchronize();    
    hipMemcpy2D(hostImg, resX * sizeof(int), devImg, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);

    for (int i = 0; i < resX * resY; ++i) 
        img[i] = hostImg[i];
    
    hipFree(devImg);
    hipHostFree(hostImg);
}
