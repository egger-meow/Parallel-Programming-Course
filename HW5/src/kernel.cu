#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int maxIterations) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < maxIterations; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandelKernel (float lowerX, float lowerY, float stepX, float stepY, int maxIterations, int* img, int resX, int resY) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < resX * resY) {
        int x = idx % resX;
        int y = idx / resX;

        float c_re = lowerX + x * stepX;
        float c_im = lowerY + y * stepY;

        int iter = mandel(c_re, c_im, maxIterations);

        img[y * resX + x] = iter;
    } 
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    size_t size = resX * resY * sizeof(int);

    int* hostImg = (int*)malloc(size);
    int* devImg;
    hipMalloc((void**)&devImg, size);
    
    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid = (resX * resY + threadsPerBlock - 1) / threadsPerBlock;

    mandelKernel<<<blocksPerGrid, threadsPerBlock>>>(
        lowerX, lowerY, stepX, stepY, maxIterations, devImg, resX, resY);

    for (int i = 0; i < resX * resY; ++i) 
        img[i] = hostImg[i];

    hipFree(devImg);
    free(devImg);
}
