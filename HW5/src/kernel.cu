#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define THREADS_PER_BLOCK 256

__device__ int mandel(float c_re, float c_im, int maxIterations) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < maxIterations; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    return i;
}

// CUDA Kernel: Each thread computes one pixel
__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, 
                             int maxIterations, int* img, size_t pitch, int resX, int resY) {
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate total number of pixels
    int totalPixels = resX * resY;

    // Ensure the thread ID is within the bounds of the image
    if (idx < totalPixels) {
        int x = idx % resX;      // Column index
        int y = idx / resX;      // Row index

        // Map pixel position to complex plane
        float c_re = lowerX + x * stepX;
        float c_im = lowerY + y * stepY;

        // Compute Mandelbrot iterations
        int iter = mandel(c_re, c_im, maxIterations);

        // Calculate the address considering the pitch
        int* row = (int*)((char*)img + y * pitch);
        row[x] = iter;
    }
}

// Host front-end function that allocates memory and launches the kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, 
            int* img, int resX, int resY, int maxIterations) {
    // Calculate step sizes
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Calculate the total number of pixels
    size_t totalPixels = resX * resY;

    // Define block and grid sizes
    int threadsPerBlockLocal = THREADS_PER_BLOCK;
    int blocksPerGrid = (totalPixels + threadsPerBlockLocal - 1) / threadsPerBlockLocal;

    // Allocate pinned host memory using cudaHostAlloc
    int* host_img;
    hipError_t err = hipHostAlloc((void**)&host_img, resX * resY * sizeof(int), hipHostMallocDefault);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate pinned host memory (error code %s)\n", 
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate pitched device memory using cudaMallocPitch
    int* dev_img;
    size_t pitch;
    err = hipMallocPitch((void**)&dev_img, &pitch, resX * sizeof(int), resY);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate pitched device memory (error code %s)\n", 
                hipGetErrorString(err));
        hipHostFree(host_img);
        exit(EXIT_FAILURE);
    }

    // Launch the kernel
    mandelKernel<<<blocksPerGrid, threadsPerBlockLocal>>>(lowerX, lowerY, stepX, stepY, 
                                                         maxIterations, dev_img, pitch, resX, resY);

    // Check for any errors during kernel launch
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch mandelKernel (error code %s)\n", 
                hipGetErrorString(err));
        hipFree(dev_img);
        hipHostFree(host_img);
        exit(EXIT_FAILURE);
    }

    // Synchronize to ensure kernel completion
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %s after launching kernel!\n", 
                hipGetErrorString(err));
        hipFree(dev_img);
        hipHostFree(host_img);
        exit(EXIT_FAILURE);
    }

    // Copy the computed image data from device to host using cudaMemcpy2D
    err = hipMemcpy2D(host_img, resX * sizeof(int), dev_img, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy image from device to host (error code %s)\n", 
                hipGetErrorString(err));
        hipFree(dev_img);
        hipHostFree(host_img);
        exit(EXIT_FAILURE);
    }

    // Copy the image data from host_img to the provided img array
    // Assuming 'img' is pre-allocated and provided by the caller
    for (int i = 0; i < resX * resY; ++i) {
        img[i] = host_img[i];
    }

    // Free device and host memory
    hipFree(dev_img);
    hipHostFree(host_img);
}