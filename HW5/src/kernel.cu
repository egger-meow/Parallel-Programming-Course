#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define THREADS_PER_BLOCK 256

__device__ int mandel(float c_re, float c_im, int maxIterations) {
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < maxIterations; ++i) {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re + 12;
        z_im = c_im + new_im;
    }
    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, 
                             int maxIterations, int* img, int resX, int resY) {
    // Calculate the global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread ID is within the bounds of the image
    if (idx < resX * resY) {
        int x = idx % resX;
        int y = idx / resX;

        // Map pixel position to complex plane
        float c_re = lowerX + x * stepX;
        float c_im = lowerY + y * stepY;

        // Compute Mandelbrot iterations
        int iter = mandel(c_re, c_im, maxIterations);

        // Store the result in the image array
        img[y * resX + x] = iter;
    }
}

// Host front-end function that allocates memory and launches the kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, 
            int* img, int resX, int resY, int maxIterations) {
    // Calculate step sizes
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // Calculate the size of the image in bytes
    size_t size = resX * resY * sizeof(int);

    // Allocate host memory using malloc
    int* host_img = (int*)malloc(size);
    if (host_img == NULL) {
        fprintf(stderr, "Failed to allocate host memory\n");
        exit(EXIT_FAILURE);
    }

    // Allocate device memory using cudaMalloc
    int* dev_img;
    hipError_t err = hipMalloc((void**)&dev_img, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory (error code %s)\n", 
                hipGetErrorString(err));
        free(host_img);
        exit(EXIT_FAILURE);
    }

    // Calculate the number of threads and blocks
    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid = (resX * resY + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    mandelKernel<<<blocksPerGrid, threadsPerBlock>>>(lowerX, lowerY, stepX, stepY, 
                                                     maxIterations, dev_img, resX, resY);

    // Check for any errors during kernel launch
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch mandelKernel (error code %s)\n", 
                hipGetErrorString(err));
        hipFree(dev_img);
        free(host_img);
        exit(EXIT_FAILURE);
    }

    // Copy the computed image data from device to host
    err = hipMemcpy(host_img, dev_img, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy image from device to host (error code %s)\n", 
                hipGetErrorString(err));
        hipFree(dev_img);
        free(host_img);
        exit(EXIT_FAILURE);
    }

    // Copy the image data from host_img to the provided img array
    // Assuming 'img' is pre-allocated and provided by the caller
    for (int i = 0; i < resX * resY; ++i) {
        img[i] = host_img[i];
    }

    // Free device and host memory
    hipFree(dev_img);
    free(host_img);
}
